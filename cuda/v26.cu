
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// #include <unistd.h>

#define TEMP_BOT 50
#define TEMP_LEFT 200
#define TEMP_UP 100
#define TEMP_RIGHT 100
#define TEMP_ARC 100
#define LENGTH_BOT 8
#define LENGTH_RIGHT 3
#define LENGTH_LEFT 6
#define LENGTH_UP 5
#define RAD_ARC 3
#define ANIMATION_FRAME_DELAY 100
#define TIME 25
#define ALFA1 1
#define ALFA2 1

#define APROX_X_NODES_NUM 125
#define THREADS_X 32
#define THREADS_Y 32



double deltaX;
double* tempCur;
double* tempNext;
double* tempCurDev;
double* tempNextDev;  
int x_nodes_num = APROX_X_NODES_NUM;
int y_nodes_num;
// __global__ int y_nodes_numDev;
double time_cur;
int arc_center_x;
int arc_center_y;
FILE *fp;
FILE * gnuplotPipe;

__device__ double fArc(double, double);
__device__ double arcCoorX(double);
__device__ double arcCoorY(double);

__global__ void Kernel(double *current, double *next, size_t x_size, size_t y_size, int arc_center_x, int arc_center_y, double dt, double deltaX) {
    unsigned int x_node = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y_node = threadIdx.y + blockIdx.y * blockDim.y;

    double d2Tdx2 = 0;
    double d2Tdy2 = 0;
    
    double x_coor = x_node * deltaX;
    double y_coor = y_node * deltaX;
    
    if ( x_node > 0 && y_node > 0 && (x_node < x_size-1) && (y_node < y_size-1) ) {

        if (!(x_node >= arc_center_x && y_node > arc_center_y && (fArc(x_coor, y_coor) >= RAD_ARC*RAD_ARC))) {
           

            if(x_node >= arc_center_x && y_node > arc_center_y) {
                
                if (fArc(x_coor + deltaX, y_coor) >= RAD_ARC*RAD_ARC) {
                    double arcX = arcCoorX(y_coor);
                    double mu = (arcX - x_coor)/deltaX;
                    d2Tdx2 = 2 * (mu*current[(x_node-1)*y_size + y_node] - (mu + 1)*current[x_node*y_size+y_node] + TEMP_ARC) / (mu*(mu+1)*deltaX*deltaX);
                    // inArc = 0;
                }
                else {
                    d2Tdx2 = (current[(x_node+1)*y_size+y_node] - 2*current[x_node*y_size+y_node] + current[(x_node-1)*y_size+y_node])/(deltaX*deltaX);
                }

                if (fArc(x_coor, y_coor+deltaX) >= RAD_ARC*RAD_ARC) {
                    double arcY = arcCoorY(x_coor);
                    double lambda = (arcY - y_coor)/deltaX;
                    d2Tdy2 = 2 * (lambda*current[x_node*y_size + y_node - 1] - (lambda+1)*current[x_node*y_size + y_node] + TEMP_ARC) / (lambda*(lambda+1)*deltaX*deltaX);
                }
                else {
                    d2Tdy2 = (current[x_node*y_size+y_node+1] - 2*current[x_node*y_size+y_node] + current[x_node*y_size+y_node-1])/(deltaX*deltaX);
                }
            }
            else {
                d2Tdx2 = (current[(x_node+1)*y_size + y_node] - 2*current[x_node*y_size + y_node] + current[(x_node-1)*y_size+y_node])/(deltaX*deltaX);                    
                d2Tdy2 = (current[x_node*y_size+y_node+1] - 2*current[x_node*y_size+y_node] + current[x_node*y_size+y_node-1])/(deltaX*deltaX);
            }

            next[x_node*y_size+y_node] = dt*(ALFA1*d2Tdx2 + ALFA2*d2Tdy2) + current[x_node*y_size+y_node];
            // next[idx_X*y_size+idx_Y] = 1.0;//20.+X[idx_X]*X[idx_X]+Y[idx_Y]*Y[idx_Y]-10.*(__cosf(2.*M_PI*X[idx_X]) + __cosf(2.*M_PI*Y[idx_Y]));
        }
    }
}


// void testPrint() {
//     for (int j = y_nodes_num - 1; j >= 0 ; j--) {
//         for (int i = 0; i < x_nodes_num; i++)
//             printf("%10lf ", tempCur[i][j]);
//         printf("\n");
//     }
//     printf("\n");
//     for (int i = 0; i < x_nodes_num; i++) {
//         for (int j = 0; j < y_nodes_num; j++)
//             printf("%lf ", tempNext[i][j]);
//         printf("\n");
//     }
// }

void printNext() {
    for (int j = y_nodes_num - 1; j >= 0 ; j--) {
        for (int i = 0; i < x_nodes_num; i++)
            fprintf(fp, "%10lf ", tempNext[i*y_nodes_num+j]);
        fprintf(fp, "\n");
    }
    fprintf(fp, "\n");
}

void printNext_Field() {
    for (int j = y_nodes_num - 1; j >= 0 ; j--) {
        for (int i = 0; i < x_nodes_num; i++)
            if (tempNext[i*y_nodes_num+j] == 1.0)
                fprintf(fp, "%d;%d ", i, j);
            else
                fprintf(fp, "*;*  ");
        fprintf(fp, "\n");
    }
    fprintf(fp, "\n");
}

void printCur() {
    for (int j = y_nodes_num - 1; j >= 0 ; j--) {
        for (int i = 0; i < x_nodes_num; i++)
            fprintf(fp, "%10lf ", tempCur[i*y_nodes_num+j]);
        fprintf(fp, "\n");
    }
    fprintf(fp, "\n");
}

// void printToGnuplot(double** temp) {
//     for(int i = 0; i < x_nodes_num; i++) {
//         for(int j = 0; j < y_nodes_num; j++) {
//             double xCoor = i*deltaX;
//             double y_coor = j*deltaX;
//             fprintf(gnuplotPipe, "%lf %lf %lf\n", xCoor, y_coor, temp[i][j]);
//         }
//         fprintf(gnuplotPipe, "\n");
//     }
//     fprintf(gnuplotPipe, "e\n");    
// }

void swapRes() {
    double* temp;
    temp = tempCur;
    tempCur = tempNext;
    tempNext = temp;
}

void swapResDev() {
    double* temp;
    temp = tempCurDev;
    tempCurDev = tempNextDev;
    tempNextDev = temp;
}

void initGranUsl(double* temp) {
    for (int i = 1; i < x_nodes_num - 1; i++) {
        temp[i*y_nodes_num+0] = TEMP_BOT;
    }
    for (int i = 1; i < y_nodes_num - 1; i++) {
        temp[0+i] = TEMP_LEFT;
    }
    for (int i = 1; i <= LENGTH_UP*(x_nodes_num - 1)/LENGTH_BOT; i++) {
        temp[i*y_nodes_num+y_nodes_num-1] = TEMP_UP;
    }
    for (int i = 1; i <= LENGTH_RIGHT*(x_nodes_num - 1)/LENGTH_BOT; i++) {
        temp[(x_nodes_num - 1)*y_nodes_num+i] = TEMP_RIGHT;
    }
    // temp[0][y_nodes_num/2] = temp[1][y_nodes_num/2]/(1 + deltaX);
}


void memAlloc(double** temp) {
    if ((*temp = (double*)malloc(x_nodes_num*y_nodes_num * sizeof(double))) == NULL) {
        printf("Не хватает памяти\n");
        exit(1);
    }
    for (unsigned int i = 0; i < x_nodes_num*y_nodes_num; i++) {
        (*temp)[i] = 0.0;
    }
    // for (int i = 0; i < x_nodes_num; i++) {
    //     if (((*temp)[i] = (double*)calloc(y_nodes_num, sizeof(double))) == NULL) {
    //         printf("Не хватает памяти\n");
    //         exit(1);
    //     }
    // }
}

__device__ double fArc(double x, double y) {
    double res = (x - LENGTH_UP)*(x - LENGTH_UP) + (y - LENGTH_RIGHT)*(y - LENGTH_RIGHT);
    return res;
}

__device__ double arcCoorX(double y) {
    double x = sqrt(RAD_ARC*RAD_ARC - (y - LENGTH_RIGHT)*(y - LENGTH_RIGHT)) + 5;
    return x;
}

__device__ double arcCoorY(double x) {
    double y = sqrt(RAD_ARC*RAD_ARC - (x - LENGTH_UP)*(x - LENGTH_UP)) + 3;
    return y;
}

int main(int argc, char** argv) {
    hipError_t cudaStatus;
	hipEvent_t GPUstart, GPUstop;
	float GPUtime = 0.0f;
    
    x_nodes_num -= (x_nodes_num - 1) % LENGTH_BOT;
    y_nodes_num = (x_nodes_num - 1) * LENGTH_LEFT/LENGTH_BOT + 1;
    
    // cudaMemcpy(&y_nodes_numDev, &y_nodes_num, 1, cudaMemcpyHostToDevice);
    // y_nodes_numDev = y_nodes_num;
    cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess) {
        printf("Last error: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }
    printf("y_nodes\n");

    if (x_nodes_num < 9) {
        printf("nodes number should be more\n");
        exit(0);
    }
    deltaX = (double)LENGTH_BOT / (x_nodes_num - 1);

    fp = fopen("result_cuda.txt", "w");
    if (fp==NULL)
        printf("Open failed\n");

    memAlloc(&tempCur);
    memAlloc(&tempNext);
    initGranUsl(tempCur);
    initGranUsl(tempNext);
    
	hipEventCreate(&GPUstart);
	hipEventCreate(&GPUstop);

	hipEventRecord(GPUstart, 0);

    printf("=====");
    hipMalloc((void**)&tempCurDev, x_nodes_num*y_nodes_num * sizeof(double));
    printf("Mem = %ld\n", x_nodes_num*y_nodes_num * sizeof(double));
    cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess) {
        printf("Last error: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }
    printf("malloc 1");
    hipMalloc((void**)&tempNextDev, x_nodes_num*y_nodes_num * sizeof(double));
    cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess) {
        printf("Last error: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }
    printf("malloc 2");
    hipMemcpy(tempCurDev, tempCur, x_nodes_num*y_nodes_num*sizeof(double), hipMemcpyHostToDevice);
    cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess) {
        printf("Last error: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }
    printf("memcpy 1");
    hipMemcpy(tempNextDev, tempCurDev, x_nodes_num*y_nodes_num*sizeof(double), hipMemcpyDeviceToDevice);
    cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess) {
        printf("Last error: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }
    printf("memcpy 2\n");
    
    printf("x = %d; y = %d\n", x_nodes_num, y_nodes_num);

    int blocks_X, blocks_Y;

	if ((x_nodes_num % THREADS_X) == 0) {
		blocks_X = (x_nodes_num / THREADS_X);
	}
	else {
		blocks_X = (x_nodes_num / THREADS_X) + 1;
    }
    if ((y_nodes_num % THREADS_Y) == 0) {
		blocks_Y = (y_nodes_num / THREADS_Y);
	}
	else {
		blocks_Y = (y_nodes_num / THREADS_Y) + 1;
	}

    dim3 N_Block (THREADS_X, THREADS_Y, 1);
    dim3 N_Grid (blocks_X, blocks_Y, 1);

    arc_center_x = (x_nodes_num - 1) * LENGTH_UP/LENGTH_BOT;
    arc_center_y = (y_nodes_num - 1) * LENGTH_RIGHT/LENGTH_LEFT;


    printCur();
    time_cur = 0;
    int krt = (x_nodes_num-1)/LENGTH_BOT;
    double dt = deltaX/(krt*krt*10.0);
    // gnuplotPipe = popen("gnuplot -persistent", "w");
    // fprintf(gnuplotPipe, "set terminal gif animate delay %d\n", ANIMATION_FRAME_DELAY);
    // fprintf(gnuplotPipe, "set output 'animate.gif'\n");
    // fprintf(gnuplotPipe, "set pm3d map interpolate 0,0\n");
    // fprintf(gnuplotPipe, "set palette defined (0 'white', 0.01 '#002137', %d '#8b00ff', %d '#8b0000', %d 'yellow')\n", TEMP_LEFT/4, TEMP_LEFT/2, TEMP_LEFT);
    // int iterCount = (int)(TIME/dt);
    // fprintf(gnuplotPipe, "do for [i=1:%d] {\n", iterCount+1);       
    // fprintf(gnuplotPipe, "splot '-' with pm3d\n");
    // fprintf(gnuplotPipe, "}\n");
    double nextFrameTime = ANIMATION_FRAME_DELAY/100;
    // printToGnuplot(tempCur);
    while (time_cur <= TIME) {

        Kernel <<< N_Grid, N_Block >>> (tempCurDev, tempNextDev, x_nodes_num, y_nodes_num, arc_center_x, arc_center_y, dt, deltaX); //////////////////
        cudaStatus = hipGetLastError();
        if(cudaStatus != hipSuccess) {
            printf("Last error: %s\n", hipGetErrorString(cudaStatus));
            return 0;
        }

        // tempNext[0][y_nodes_num/2] = tempNext[1][y_nodes_num/2]/(1 + deltaX);        
        if(time_cur >= nextFrameTime) {
            hipMemcpy(tempNext, tempNextDev, x_nodes_num*y_nodes_num*sizeof(double), hipMemcpyDeviceToHost);
            // printf("%lf\n", tempNext[0]);
            printNext();
            // printNext_Field();
            // printToGnuplot(tempNext);
            // printf("Time: %lf\t0 - %lf   1 - %lf\n",time, tempNext[0][y_nodes_num/2], tempNext[1][y_nodes_num/2]);
            nextFrameTime += (double)ANIMATION_FRAME_DELAY/100;
        }
        // printf("%lf\n", time);
        time_cur += dt;
        swapResDev();
    }

    hipEventRecord(GPUstop, 0);
	hipEventSynchronize(GPUstop);

	hipEventElapsedTime(&GPUtime, GPUstart, GPUstop);
	printf("GPU time : %.3f ms\n", GPUtime);

    // swapRes();
    // testPrint();

    // pclose(gnuplotPipe);
    fclose(fp);
    return 0;
}

// for (int y_node = 1; y_node < y_nodes_num - 1; y_node++) {
//     int inArc = 1;
//     for (int x_node = 1; (x_node < x_nodes_num - 1) && inArc; x_node++) {
//         if(x_node >= arc_center_x && y_node > arc_center_y) {
//             double x_coor = x_node * deltaX;
//             double y_coor = y_node * deltaX;
            
//             if (fArc(x_coor + deltaX, y_coor) >= RAD_ARC*RAD_ARC) {
//                 double arcX = arcCoorX(y_coor);
//                 double mu = (arcX - x_coor)/deltaX;
//                 d2Tdx2 = 2 * (mu*tempCur[(x_node-1)*y_nodes_num + y_node] - (mu + 1)*tempCur[x_node*y_nodes_num+y_node] + TEMP_ARC) / (mu*(mu+1)*deltaX*deltaX);
//                 inArc = 0;
//             }
//             else {
//                 d2Tdx2 = (tempCur[(x_node+1)*y_nodes_num+y_node] - 2*tempCur[x_node*y_nodes_num+y_node] + tempCur[(x_node-1)*y_nodes_num+y_node])/(deltaX*deltaX);
//             }

//             if (fArc(x_coor, y_coor+deltaX) >= RAD_ARC*RAD_ARC) {
//                 double arcY = arcCoorY(x_coor);
//                 double lambda = (arcY - y_coor)/deltaX;
//                 d2Tdy2 = 2 * (lambda*tempCur[x_node*y_nodes_num + y_node - 1] - (lambda+1)*tempCur[x_node*y_nodes_num + y_node] + TEMP_ARC) / (lambda*(lambda+1)*deltaX*deltaX);
//             }
//             else {
//                 d2Tdy2 = (tempCur[x_node*y_nodes_num+y_node+1] - 2*tempCur[x_node*y_nodes_num+y_node] + tempCur[x_node*y_nodes_num+y_node-1])/(deltaX*deltaX);
//             }
//         }
//         else {
//             d2Tdx2 = (tempCur[(x_node+1)*y_nodes_num + y_node] - 2*tempCur[x_node*y_nodes_num + y_node] + tempCur[(x_node-1)*y_nodes_num+y_node])/(deltaX*deltaX);                    
//             d2Tdy2 = (tempCur[x_node*y_nodes_num+y_node+1] - 2*tempCur[x_node*y_nodes_num+y_node] + tempCur[x_node*y_nodes_num+y_node-1])/(deltaX*deltaX);
//         }

//         tempNext[x_node*y_nodes_num+y_node] = dt*(ALFA1*d2Tdx2 + ALFA2*d2Tdy2) + tempCur[x_node*y_nodes_num+y_node];
//     }
// }