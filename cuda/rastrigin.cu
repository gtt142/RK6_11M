
#include <hip/hip_runtime.h>
#include <stdio.h>

const int DIM = 2500;

__global__ void Kernel(float *X, float *Y, float *Z) {
    unsigned int idx_X = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx_Y = threadIdx.y + blockIdx.y * blockDim.y;
    // printf("%d\n", idx_X*DIM+idx_Y);
    Z[idx_X*DIM+idx_Y] = 20.+X[idx_X]*X[idx_X]+Y[idx_Y]*Y[idx_Y]-10.*(__cosf(2.*M_PI*X[idx_X]) + __cosf(2.*M_PI*Y[idx_Y]));
}

void initialization(float min, float max, float* mem, int dim) {
    float delta = (max - min) / (dim - 1);
    for (int i = 0; i < dim; i++) {
        mem[i] = min + delta * i;
    }
}

int main() {
    hipError_t cudaStatus;
    size_t mem_size = sizeof(float)*DIM;
    float *hostX, *hostY, *hostZ;
    float *devX, *devY, *devZ;
    hostX = (float*)malloc(mem_size);
    hostY = (float*)malloc(mem_size);
    hostZ = (float*)malloc(mem_size*DIM);
    hipMalloc((void**)&devX, mem_size);
    hipMalloc((void**)&devY, mem_size);
    hipMalloc((void**)&devZ, mem_size*DIM);
    initialization(-5, 5, hostX, DIM);
    hipMemcpy(devX, hostX, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(devY, devX, mem_size, hipMemcpyDeviceToDevice);
    dim3 N_Block (32, 32, 1);
    dim3 N_Grid (DIM/32,DIM/32,1);
    Kernel <<< N_Grid, N_Block >>> (devX,devY,devZ);
    hipMemcpy(hostZ, devZ, mem_size*DIM, hipMemcpyDeviceToHost);
	
    cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess) {
		printf("Last error: %s\n", hipGetErrorString(cudaStatus));
		return 0;
	}
    
    hipFree(devX);
    hipFree(devY);
    hipFree(devZ);

    for (int i = 0; i < DIM; i++) {
        for (int j = 0; j < DIM; j++) {
            printf("%lf;", hostZ[i*DIM + j]);
        }
        printf("\n");
    }

    free(hostX);
    free(hostY);
    free(hostZ);
    return 0;
}

